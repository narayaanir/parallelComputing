#include "hip/hip_runtime.h"
/*
 * File: RandomWalkGpu.cu
 * FileType: CUDA
 *
 * This file is part of the program RandomWalkGpu to find the average distance
 * of N particles performing a three dimensional Random Walk. The random walks
 * are calculated in the GPU.
 *
 * Author: Anirudh S N
 * Date: Nov 9, 2014
 *
*/
#include "Random.cu"
// USed to define the number of threads in each block
#define NT 1024
// Declaring device and block specific variable names
__device__ double devPos;
__shared__ double shrPosX[NT];
__shared__ double shrPosY[NT];
__shared__ double shrPosZ[NT];

/*
 * Method : atomicAdd
 * 
 * Description: This method is used to perform atomicAdd on two double values.
 * 
 * @Parameters
 *      v: pointer to a double variable indicating target
 *      value: double value indicating the value to be added.
 *
*/
__device__ void atomicAdd(double *v, double value) {
    double oldval, newval; 
    do {
        oldval = *v; 
        newval = oldval + value; 
    } while (atomicCAS((unsigned long long int *)v, 
        __double_as_longlong (oldval), 
        __double_as_longlong (newval)) 
        != __double_as_longlong (oldval)); 
}

/*
 * Method : RandomWalkGpu
 * 
 * Description: The kernel starts.
 * 
 * @Parameters
 *      N: indiacates the number of individual members to be calculated
 *      T: indicates the number of iterations to be performed
 *      seed: indicates the seed value for the random number generation
 *
*/
extern "C" __global__ void RandomWalkGpu 
(unsigned long long int N, 
unsigned long long int T,
unsigned long long int seed) {
    int thr, rank;
    double xCoor=0;
    double yCoor=0;
    double zCoor=0;
    double dist=0;
    prng_t prng;
    
    thr=threadIdx.x;
//  int  size=gridDim.x*NT;
    rank=blockIdx.x*NT + thr;
    prngSetSeed (&prng, seed + rank);
    


        for (unsigned long long int i=thr; i<T; i+=NT) {
            int step=prngNextInt(&prng, 6);
//            printf("==%d", step);
            if (step==0) {
                xCoor+=1;
            } else if (step==1) {
                xCoor-=1;
            } else if (step==2) {
                yCoor+=1;
            } else if (step==3) {
                yCoor-=1;
            } else if (step==4) {
                zCoor+=1;
            } else {
                zCoor-=1;
            }
        }
        shrPosX[thr]=xCoor;
        shrPosY[thr]=yCoor;
        shrPosZ[thr]=zCoor;

    __syncthreads();

    for (int i=NT/2; i>0; i>>=1) {
        if (thr<i) {
//            printf("==%d\n", i);
            shrPosX[thr]+=shrPosX[thr+i];
            shrPosY[thr]+=shrPosY[thr+i];
            shrPosZ[thr]+=shrPosZ[thr+i];
        }
        __syncthreads();
    }
    
    if (thr==0) {
//        printf("--%f %f %f\n", shrPosX[0], shrPosY[0], shrPosZ[0]);
        double xSquared, ySquared, zSquared;
        xSquared=shrPosX[0]*shrPosX[0];
        ySquared=shrPosY[0]*shrPosY[0];
        zSquared=shrPosZ[0]*shrPosZ[0];

        dist+=sqrt(xSquared+ySquared+zSquared);
        atomicAdd (&devPos, dist);      
    }

}
